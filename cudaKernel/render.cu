#include "hip/hip_runtime.h"
#include "util.h"
#include "hip/hip_vector_types.h"
#include <hip/hip_runtime.h>
#include "glm/glm.hpp"


/************************************************** ray casting functions *********************************************/
typedef struct	/* a data sturctur to hold 4x4 view matrix*/
{
    float4 m[4];
} mat4;

__constant__ mat4 d_invViewMatrix;  // inverse view matrix


struct Ray					/* data sturctur to hold our ray that will be shooting*/
{
    float3 origine;   // origin
    float3 direction;   // direction
};


// intersect ray with a box  every body using this
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm

__device__ int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
    // compute intersection of ray with all six bbox planes
    float3 invR = make_float3(1.0f) / r.direction;
    float3 tbot = invR * (boxmin - r.origine);
    float3 ttop = invR * (boxmax - r.origine);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

    *tnear = largest_tmin;
    *tfar = smallest_tmax;

    return smallest_tmax > largest_tmin;
}

__device__ float3 mul(const mat4 &M, const float3 &v)	/* this multiplacation will be used in raycasting calculation  since all calculation are using float3 and stuff and i'am to tired to change every thing to glm::vec and mats*/
{
    float3 r;
    r.x = dot(v, make_float3(M.m[0]));
    r.y = dot(v, make_float3(M.m[1]));
    r.z = dot(v, make_float3(M.m[2]));
    return r;
}

// transform vector by matrix with translation
__device__
float4 mul(const mat4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.m[0]);
    r.y = dot(v, M.m[1]);
    r.z = dot(v, M.m[2]);
    r.w = 1.0f;
    return r;
}
__device__ uint rgbaFloatToInt(float4 rgba)	/* covert float to int */
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}

/**************************************** carving functions**************************************************************/
__device__ float4 MultiplicationMatrixVector(glm::mat4 Mat, glm::vec4 Vec) /* this will be used for projection in silhouette becoz all data are using glm*/
{

	glm::vec4 w(0.0);

	w = Mat * Vec;
	float4 reslut = make_float4(w.x,w.y,w.z,w.w);
	return reslut;
}
__device__ float4 Projection(glm::mat4 Mat,float x, float y, float z)
{
	float4 screen;
	glm::vec4 Vec(x,y,z,1.0);
	screen =  MultiplicationMatrixVector(Mat,Vec);

	screen.x = screen.x / screen.z;
	screen.y = screen.y / screen.z;
	//screen.z = screen.z / screen.z;
	//screen.w = screen.w / screen.z;

	return screen;
}
__device__ int Check(uchar4 *Slihouette,int i, int w,int h,int imgnbr,int u , int v)
{
	if((u>= 0  && u < w) && (v >=0 && v< h) )
	{
		int j =(v * w )+ u;
		int imgsize = w * h;
		if(Slihouette[(i*imgsize) + j].w == 1.0)
		{
			return 1;
		}
	}
	return -1;
}

__device__ float4 blendingColor(float depth1,float depth2,float4 c1,float4 c2) // this fucntion simpley blende 2 color but instead of using alpha we use depth
{
	// this function didn't work well probalby need to be tooned in and try it with someting else
	float alpha = 1- (1-depth1) * (1-depth2);

	float r =  c1.x * depth1 / alpha + c2.x * depth2 * (1 - depth1) / alpha;
	float g =  c1.y * depth1 / alpha + c2.y * depth2 * (1 - depth1) / alpha;
	float b =  c1.z * depth1 / alpha + c2.z * depth2 * (1 - depth1) / alpha;
	float a = 1.0;

	return make_float4(r,g,b,a);
}
__device__ float4 lerping(float t, float4 c1,float4 c2)// c(t) = (1 − t)c 1 + tc 2 ,t ∈ [0, 1] linear interpolation
{
	float invt = 1-t;

	float r =  invt  * c1.x + t * c2.x;
	float g =  invt  * c1.y + t * c2.y;
	float b =  invt  * c1.z + t * c2.z;
	float a =  invt  * c1.w + t * c2.w;

	return make_float4(r,g,b,a);
}

// i will use the availbal code from Nvidia samples next time when i will try to optimise i will change stuff
__global__ void raycaster(uint *d_output,hipTextureObject_t texObject,glm::vec3 vCamPosition, int imageW, int imageH/*,unsigned char *slihouette*/,
						  glm::mat4 *camera, uchar4* images,bool isCOlor,unsigned int imageNumber, int w, int h)
{

    const int maxSteps = 1000;
    const float tstep = 0.01f;
    const float3 boxMin = make_float3(-1.0f, -1.0f, -0.55f); // lazem nahder 3liha fi papier achffa 3laahe ykrojo double  genre ranna ndorro dakle cycle par example l'intersection (0.13,0.5,0.5) ta3ti nafsse texel li tamdo l'intersection (1.0,2.0,1.5) XD
    const float3 boxMax = make_float3(1.0f, 1.0f, 0.55f);
   /* const float3 lightPosition = make_float3(150.0f,150.0f,0.0f); // this should be declared out of here  it shoud be passed as render parameter
    const float4 lightCOlor    = make_float4(1.0f,1.0f,1.0f,1.0f);*/
    float depth = 10000.0f;
  //  float3 gradient;
   // float3 lightDirection;

    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    if ((x >= imageW) || (y >= imageH)) return;
    d_output[y*imageW + x]  = rgbaFloatToInt(make_float4(0.1,0.1,0.75,1.0));


    float u = (x / (float) imageW)*2.0f-1.0f;
    float v = (y / (float) imageH)*2.0f-1.0f;

    // calculate eye ray in world space
    Ray eyeRay;
    eyeRay.origine = make_float3(mul(d_invViewMatrix, make_float4(vCamPosition.x,vCamPosition.y,vCamPosition.z, 1.0f)));  // use this to move the camera not the object make_float3( vCamPosition.x,vCamPosition.y,vCamPosition.z);
    eyeRay.direction = normalize(make_float3(u, v, -2.0f));
    eyeRay.direction = mul(d_invViewMatrix, eyeRay.direction);

    // find intersection with box
    float tnear, tfar;
    int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

    if (!hit) return;

    if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

    // march along ray from front to back, accumulating color
    float t = tnear;
    float3 pos = eyeRay.origine + eyeRay.direction *tnear;
    float3 step = eyeRay.direction * tstep;

    float4 rgbaFinale = make_float4(0.0f,0.0f,0.0f,1.0f);
    for (int i=0; i<maxSteps; i++)
    {
        // read from 3D texture
        // remap position to [0, 1] coordinates
        float4 texel = tex3D<float4>(texObject,pos.x*0.5f+0.5f, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);
        // now that we have the texel wich represent the position of volume we wanna carve let's do the silhouette consistence test

        bool InAll;
		for(int j = 0; j< imageNumber; j++)
		{

			float4 screen  = Projection(camera[j],texel.x,  texel.y, texel.z);
			int u = (int)screen.x;
			int v = (int)screen.y;
			int check = Check(images,j,w,h,imageNumber, u , v);
			// calulate color whever  it's correct or not

			if(check == 1)
			{
				InAll = true;

				int indexer =(v * w)+ u;
				if(screen.z < depth)
					{
						float4 col = make_float4(  (float)images[(j*w*h) + indexer].x / 255.0
												  ,(float)images[(j*w*h) + indexer].y / 255.0
												  ,(float)images[(j*w*h) + indexer].z / 255.0
												 ,1.0);
							rgbaFinale = lerping(0.5,rgbaFinale,col); //lerp hurt the correct and almost correct the wrong
							depth = screen.z;
					}
			}
			else
			{
				depth = 10000.0f;
				InAll = false;
				break;
			}
		}
		if(InAll == true)
		{
			/*if(isCOlor == false)
			{
				// just give one color
				float4 rgba =  1.5 *  make_float4(0.5f,0.5f,0.5f,1.0f);
				d_output[y*imageW + x]  = rgbaFloatToInt(rgba);
				return;
			}*/
			//else
			//{

				rgbaFinale = 1.5 * (rgbaFinale); // divding the lerped color by image number gived somthing corret
				d_output[y*imageW + x]  = rgbaFloatToInt(rgbaFinale );
				return;
			//}
		 }
		/*else
		{
		    float4 rgba = make_float4(0.5,0.5,0.5,0.3);
		    d_output[y*imageW + x]  = rgbaFloatToInt(rgba);
		}*/
        t += tstep;
        /*
         * TO DO add an early termination
         */
        if (t > tfar)
        	return;
        pos += step;
    }

}


extern "C"
void CallKernelraycaster(uint *d_output,hipTextureObject_t texObject,glm::vec3 vCamPosition, int imageW, int imageH/*,unsigned char *slihouette*/,glm::mat4 *camera, uchar4* images,bool isCOlor,unsigned int imageNumber,int w, int h,dim3 Blocks,dim3 Threads )
{
	hipError_t err ;
	raycaster<<<Blocks,Threads>>>(d_output,texObject,vCamPosition, imageW, imageH/*,slihouette*/,camera,images,isCOlor,imageNumber,w,h);
	err = hipGetLastError();
	if ( hipSuccess != err)
		fprintf(stderr, "Failed (error code %s)!\n", hipGetErrorString(err));
}

extern "C"
void copyInvViewMatrix(const float* invViewMatrix, size_t sizeofMatrix)
{
	CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(d_invViewMatrix), invViewMatrix, sizeofMatrix));
}

