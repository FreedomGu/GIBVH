#include "hip/hip_runtime.h"
/****************************************************************************
 * Author : saouli abdelhak (Hakosoft)
 * date : 11/07/2015
 *
 * this kernel is for to find a silhouette from an image its' simple task
 * we can add complicated stuff later on
 ****************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "util.h"

__device__ int ThreadID_2D_2D(unsigned int width)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	return y * width + x;
}

// this kernel is used for a specific data set (dinausre) once i start using camera i will write one kernel

__global__ void rgbaToSilhouette(uchar4* input,int CameraID,uchar4* ImageSilhouettes/*,unsigned char* output*/, unsigned int rows, unsigned int cols)
{

	int thread_1D_pos = ThreadID_2D_2D(cols);
	uchar4 rgba;
	rgba = input[thread_1D_pos];
	int imageSize = rows * cols;
	if(rgba.x>rgba.z)
	{
		rgba.w = 1.0;
		ImageSilhouettes[(CameraID*imageSize) + thread_1D_pos] = rgba;
	}
		//output[thread_1D_pos] = 255;
	else
	{
		rgba.w = 0.0;
		ImageSilhouettes[(CameraID*imageSize) + thread_1D_pos] = rgba;
	}
		//output[thread_1D_pos] = 0;
}

//use for other data sets
__global__ void rgbaToSilhouette2(uchar4* input,int CameraID,uchar4* ImageSilhouettes/*, unsigned char* output*/, unsigned int rows, unsigned int cols) //this white background
{

	int thread_1D_pos = ThreadID_2D_2D(cols);
	if(thread_1D_pos > rows * cols)
		return;
	uchar4 rgba;
	int imageSize = rows * cols;

	rgba = input[thread_1D_pos];
	float grayIn = .299f * rgba.x + .587f * rgba.y + .114f * rgba.z;
	if(grayIn >= 200.0)
	{
		//output[thread_1D_pos] = 0;
		rgba.w = 0.0;
		ImageSilhouettes[(CameraID*imageSize) + thread_1D_pos] = rgba;
	}

	else
	{
		//output[thread_1D_pos] = 255;
		rgba.w = 1.0;
		ImageSilhouettes[(CameraID*imageSize) + thread_1D_pos] = rgba;
	}

}


__global__ void rgbaToSilhouette3(uchar4* input,int CameraID,uchar4* ImageSilhouettes/*, unsigned char* output*/, unsigned int rows, unsigned int cols) //this white background
{

	int thread_1D_pos = ThreadID_2D_2D(cols);
	if(thread_1D_pos > rows * cols)
		return;
	uchar4 rgba;
	int imageSize = rows * cols;

	rgba = input[thread_1D_pos];

	float grayIn = .299f * rgba.z + .587f * rgba.y + .114f * rgba.x;
	if(grayIn > 20.0)
	{
		/*input[thread_1D_pos].x = 255;
		input[thread_1D_pos].y = 255;
		input[thread_1D_pos].z = 255;
		input[thread_1D_pos].w = 255;*/
		rgba.w = 1.0;
		ImageSilhouettes[(CameraID*imageSize) + thread_1D_pos] = rgba;
	}

	else
	{
		/*input[thread_1D_pos].x = 0.0;
		input[thread_1D_pos].y = 0.0;
		input[thread_1D_pos].z = 0.0;
		input[thread_1D_pos].w = 255;*/
		rgba.w = 0.0;
		ImageSilhouettes[(CameraID*imageSize) + thread_1D_pos] = rgba;
	}

}
extern "C"
void CallBackrgbaToSilhouetteKernel(uchar4* input,int CameraID,uchar4* ImageSilhouettes/*, unsigned char* output*/, unsigned int rows, unsigned int cols,
							        dim3 blocks,dim3 grid)
{
	rgbaToSilhouette<<<grid,blocks>>>(input,CameraID,ImageSilhouettes/*,output*/,rows,cols);
	hipDeviceSynchronize();
	CUDA_CHECK_RETURN(hipGetLastError());
}
